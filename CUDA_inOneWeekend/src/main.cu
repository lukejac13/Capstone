#include "hip/hip_runtime.h"
#include "rtweekend.h"

#include <chrono>
#include <iostream>

#include "camera.h"
#include "hittable.h"
#include "hittable_list.h"
#include "material.h"
#include "sphere.h"
#include "rect.h"
#include "box.h"




int main() {
    hittable_list world;

    // Create the ground/street
    auto street_material = make_shared<lambertian>(color(0.3, 0.3, 0.3));
    world.add(make_shared<sphere>(point3(0,-1000,0), 1000, street_material));

    // Define some materials for different building types
    auto concrete_material = make_shared<lambertian>(color(0.6, 0.6, 0.6));
    auto brick_material = make_shared<lambertian>(color(0.7, 0.4, 0.3));
    auto glass_material = make_shared<dielectric>(1.5);
    auto metal_material = make_shared<metal>(color(0.8, 0.8, 0.9), 0.1);
    auto blue_building = make_shared<metal>(color(0.2, 0.4, 0.8), 0.1);
    auto red_building = make_shared<lambertian>(color(0.8, 0.3, 0.2));
    auto green_building = make_shared<metal>(color(0.3, 0.7, 0.4), 0.1);

    // Create a city grid of buildings
    
    // Skyscraper 1 - Tall glass tower
    world.add(make_shared<box>(point3(-8, 0, -8), point3(-6, 12, -6), glass_material));
    
    // Skyscraper 2 - Metal and concrete tower
    world.add(make_shared<box>(point3(6, 0, -8), point3(8, 15, -6), metal_material));
    
    // Skyscraper 3 - Very tall center building
    world.add(make_shared<box>(point3(-1, 0, -1), point3(1, 18, 1), concrete_material));
    
    // Medium height buildings
    world.add(make_shared<box>(point3(-8, 0, 2), point3(-6, 8, 4), brick_material));
    world.add(make_shared<box>(point3(-4, 0, -8), point3(-2, 6, -6), blue_building));
    world.add(make_shared<box>(point3(2, 0, -8), point3(4, 7, -6), red_building));
    world.add(make_shared<box>(point3(6, 0, 2), point3(8, 9, 4), green_building));
    world.add(make_shared<box>(point3(-8, 0, 6), point3(-6, 5, 8), concrete_material));
    
    // Low-rise buildings
    world.add(make_shared<box>(point3(-4, 0, 2), point3(-2, 4, 4), brick_material));
    world.add(make_shared<box>(point3(2, 0, 2), point3(4, 3, 4), blue_building));
    world.add(make_shared<box>(point3(-4, 0, 6), point3(-2, 4, 8), red_building));
    world.add(make_shared<box>(point3(2, 0, 6), point3(4, 5, 8), green_building));
    
    // Create some stepped buildings (more complex structures)
    // Building with setbacks
    world.add(make_shared<box>(point3(-12, 0, -4), point3(-10, 8, -2), concrete_material));
    world.add(make_shared<box>(point3(-11.5, 8, -3.5), point3(-10.5, 12, -2.5), concrete_material));
    world.add(make_shared<box>(point3(-11.25, 12, -3.25), point3(-10.75, 15, -2.75), concrete_material));
    
    // L-shaped building
    world.add(make_shared<box>(point3(10, 0, -4), point3(12, 6, -2), brick_material));
    world.add(make_shared<box>(point3(10, 0, -2), point3(14, 6, 0), brick_material));
    
    // Add some architectural details - rooftop structures
    world.add(make_shared<box>(point3(-0.5, 18, -0.5), point3(0.5, 20, 0.5), metal_material)); // Antenna/spire
    world.add(make_shared<box>(point3(6.5, 15, -7.5), point3(7.5, 17, -6.5), glass_material)); // Rooftop structure
    
    // Add glass windows to all buildings
    auto window_glass = make_shared<dielectric>(1.5);
    
    // Windows for Skyscraper 1 (Glass tower at -8 to -6, height 12)
    // Since it's already glass, add some window frames with metal
    auto window_frame = make_shared<metal>(color(0.3, 0.3, 0.3), 0.0);
    for (int floor = 1; floor < 12; floor += 2) {
        // Front face windows
        world.add(make_shared<yz_rect>(floor, floor + 1.5, -7.8, -6.2, -8.01, window_glass));
        // Back face windows  
        world.add(make_shared<yz_rect>(floor, floor + 1.5, -7.8, -6.2, -5.99, window_glass));
        // Side face windows
        world.add(make_shared<xy_rect>(-7.8, -6.2, floor, floor + 1.5, -8.01, window_glass));
        world.add(make_shared<xy_rect>(-7.8, -6.2, floor, floor + 1.5, -5.99, window_glass));
    }
    
    // Windows for Skyscraper 2 (Metal tower at 6 to 8, height 15)
    for (int floor = 1; floor < 15; floor += 2) {
        // Front face windows
        world.add(make_shared<yz_rect>(floor, floor + 1.5, -7.8, -6.2, 5.99, window_glass));
        // Back face windows
        world.add(make_shared<yz_rect>(floor, floor + 1.5, -7.8, -6.2, 8.01, window_glass));
        // Side face windows
        world.add(make_shared<xy_rect>(6.2, 7.8, floor, floor + 1.5, -8.01, window_glass));
        world.add(make_shared<xy_rect>(6.2, 7.8, floor, floor + 1.5, -5.99, window_glass));
    }
    
    // Windows for Skyscraper 3 (Central tower at -1 to 1, height 18)
    for (int floor = 1; floor < 18; floor += 2) {
        // Front face windows
        world.add(make_shared<yz_rect>(floor, floor + 1.5, -0.8, 0.8, -1.01, window_glass));
        // Back face windows
        world.add(make_shared<yz_rect>(floor, floor + 1.5, -0.8, 0.8, 1.01, window_glass));
        // Side face windows
        world.add(make_shared<xy_rect>(-0.8, 0.8, floor, floor + 1.5, -1.01, window_glass));
        world.add(make_shared<xy_rect>(-0.8, 0.8, floor, floor + 1.5, 1.01, window_glass));
    }
    
    // Windows for medium height buildings
    // Building at (-8, 0, 2) to (-6, 8, 4) - brick material
    for (int floor = 1; floor < 8; floor += 2) {
        world.add(make_shared<yz_rect>(floor, floor + 1.5, 2.2, 3.8, -8.01, window_glass));
        world.add(make_shared<yz_rect>(floor, floor + 1.5, 2.2, 3.8, -5.99, window_glass));
        world.add(make_shared<xy_rect>(-7.8, -6.2, floor, floor + 1.5, 1.99, window_glass));
        world.add(make_shared<xy_rect>(-7.8, -6.2, floor, floor + 1.5, 4.01, window_glass));
    }
    
    // Building at (-4, 0, -8) to (-2, 6, -6) - blue building
    for (int floor = 1; floor < 6; floor += 2) {
        world.add(make_shared<yz_rect>(floor, floor + 1.5, -7.8, -6.2, -4.01, window_glass));
        world.add(make_shared<yz_rect>(floor, floor + 1.5, -7.8, -6.2, -1.99, window_glass));
        world.add(make_shared<xy_rect>(-3.8, -2.2, floor, floor + 1.5, -8.01, window_glass));
        world.add(make_shared<xy_rect>(-3.8, -2.2, floor, floor + 1.5, -5.99, window_glass));
    }
    
    // Building at (2, 0, -8) to (4, 7, -6) - red building
    for (int floor = 1; floor < 7; floor += 2) {
        world.add(make_shared<yz_rect>(floor, floor + 1.5, -7.8, -6.2, 1.99, window_glass));
        world.add(make_shared<yz_rect>(floor, floor + 1.5, -7.8, -6.2, 4.01, window_glass));
        world.add(make_shared<xy_rect>(2.2, 3.8, floor, floor + 1.5, -8.01, window_glass));
        world.add(make_shared<xy_rect>(2.2, 3.8, floor, floor + 1.5, -5.99, window_glass));
    }
    
    // Building at (6, 0, 2) to (8, 9, 4) - green building
    for (int floor = 1; floor < 9; floor += 2) {
        world.add(make_shared<yz_rect>(floor, floor + 1.5, 2.2, 3.8, 5.99, window_glass));
        world.add(make_shared<yz_rect>(floor, floor + 1.5, 2.2, 3.8, 8.01, window_glass));
        world.add(make_shared<xy_rect>(6.2, 7.8, floor, floor + 1.5, 1.99, window_glass));
        world.add(make_shared<xy_rect>(6.2, 7.8, floor, floor + 1.5, 4.01, window_glass));
    }
    
    // Building at (-8, 0, 6) to (-6, 5, 8) - concrete
    for (int floor = 1; floor < 5; floor += 2) {
        world.add(make_shared<yz_rect>(floor, floor + 1.5, 6.2, 7.8, -8.01, window_glass));
        world.add(make_shared<yz_rect>(floor, floor + 1.5, 6.2, 7.8, -5.99, window_glass));
        world.add(make_shared<xy_rect>(-7.8, -6.2, floor, floor + 1.5, 5.99, window_glass));
        world.add(make_shared<xy_rect>(-7.8, -6.2, floor, floor + 1.5, 8.01, window_glass));
    }
    
    // Windows for low-rise buildings
    // Building at (-4, 0, 2) to (-2, 4, 4) - brick
    for (int floor = 1; floor < 4; floor += 2) {
        world.add(make_shared<yz_rect>(floor, floor + 1.5, 2.2, 3.8, -4.01, window_glass));
        world.add(make_shared<yz_rect>(floor, floor + 1.5, 2.2, 3.8, -1.99, window_glass));
        world.add(make_shared<xy_rect>(-3.8, -2.2, floor, floor + 1.5, 1.99, window_glass));
        world.add(make_shared<xy_rect>(-3.8, -2.2, floor, floor + 1.5, 4.01, window_glass));
    }
    
    // Building at (2, 0, 2) to (4, 3, 4) - blue
    for (int floor = 1; floor < 3; floor += 2) {
        world.add(make_shared<yz_rect>(floor, floor + 1.5, 2.2, 3.8, 1.99, window_glass));
        world.add(make_shared<yz_rect>(floor, floor + 1.5, 2.2, 3.8, 4.01, window_glass));
        world.add(make_shared<xy_rect>(2.2, 3.8, floor, floor + 1.5, 1.99, window_glass));
        world.add(make_shared<xy_rect>(2.2, 3.8, floor, floor + 1.5, 4.01, window_glass));
    }
    
    // Building at (-4, 0, 6) to (-2, 4, 8) - red
    for (int floor = 1; floor < 4; floor += 2) {
        world.add(make_shared<yz_rect>(floor, floor + 1.5, 6.2, 7.8, -4.01, window_glass));
        world.add(make_shared<yz_rect>(floor, floor + 1.5, 6.2, 7.8, -1.99, window_glass));
        world.add(make_shared<xy_rect>(-3.8, -2.2, floor, floor + 1.5, 5.99, window_glass));
        world.add(make_shared<xy_rect>(-3.8, -2.2, floor, floor + 1.5, 8.01, window_glass));
    }
    
    // Building at (2, 0, 6) to (4, 5, 8) - green
    for (int floor = 1; floor < 5; floor += 2) {
        world.add(make_shared<yz_rect>(floor, floor + 1.5, 6.2, 7.8, 1.99, window_glass));
        world.add(make_shared<yz_rect>(floor, floor + 1.5, 6.2, 7.8, 4.01, window_glass));
        world.add(make_shared<xy_rect>(2.2, 3.8, floor, floor + 1.5, 5.99, window_glass));
        world.add(make_shared<xy_rect>(2.2, 3.8, floor, floor + 1.5, 8.01, window_glass));
    }
    
    // Windows for stepped building (-12, 0, -4) to (-10, 8, -2)
    for (int floor = 1; floor < 8; floor += 2) {
        world.add(make_shared<yz_rect>(floor, floor + 1.5, -3.8, -2.2, -12.01, window_glass));
        world.add(make_shared<yz_rect>(floor, floor + 1.5, -3.8, -2.2, -9.99, window_glass));
        world.add(make_shared<xy_rect>(-11.8, -10.2, floor, floor + 1.5, -4.01, window_glass));
        world.add(make_shared<xy_rect>(-11.8, -10.2, floor, floor + 1.5, -1.99, window_glass));
    }
    
    // Windows for L-shaped building part 1: (10, 0, -4) to (12, 6, -2)
    for (int floor = 1; floor < 6; floor += 2) {
        world.add(make_shared<yz_rect>(floor, floor + 1.5, -3.8, -2.2, 9.99, window_glass));
        world.add(make_shared<yz_rect>(floor, floor + 1.5, -3.8, -2.2, 12.01, window_glass));
        world.add(make_shared<xy_rect>(10.2, 11.8, floor, floor + 1.5, -4.01, window_glass));
        world.add(make_shared<xy_rect>(10.2, 11.8, floor, floor + 1.5, -1.99, window_glass));
    }
    
    // Windows for L-shaped building part 2: (10, 0, -2) to (14, 6, 0)
    for (int floor = 1; floor < 6; floor += 2) {
        world.add(make_shared<yz_rect>(floor, floor + 1.5, -1.8, -0.2, 9.99, window_glass));
        world.add(make_shared<yz_rect>(floor, floor + 1.5, -1.8, -0.2, 14.01, window_glass));
        world.add(make_shared<xy_rect>(10.2, 13.8, floor, floor + 1.5, -2.01, window_glass));
        world.add(make_shared<xy_rect>(10.2, 13.8, floor, floor + 1.5, 0.01, window_glass));
    }
    
    // Create roads/streets using rectangles
    auto road_material = make_shared<lambertian>(color(0.2, 0.2, 0.2));
    
    // Main street running north-south
    world.add(make_shared<xz_rect>(-0.5, 0.5, -15, 15, 0.01, road_material));
    
    // Cross street running east-west
    world.add(make_shared<xz_rect>(-15, 15, -0.5, 0.5, 0.01, road_material));
    
    // Add some smaller streets
    world.add(make_shared<xz_rect>(-5.5, -4.5, -15, 15, 0.01, road_material));
    world.add(make_shared<xz_rect>(4.5, 5.5, -15, 15, 0.01, road_material));
    world.add(make_shared<xz_rect>(-15, 15, -5.5, -4.5, 0.01, road_material));
    world.add(make_shared<xz_rect>(-15, 15, 4.5, 5.5, 0.01, road_material));
    
    // Add some decorative elements - floating glass panels (like billboards)
    auto billboard_material = make_shared<dielectric>(1.3);
    world.add(make_shared<yz_rect>(5, 7, -3, -1, -9, billboard_material));
    world.add(make_shared<xy_rect>(-3, -1, 8, 10, 9, billboard_material));
    
    // Add some spherical elements (could be decorative or water towers)
    auto water_tower = make_shared<metal>(color(0.7, 0.7, 0.8), 0.2);
    world.add(make_shared<sphere>(point3(-7, 13, -7), 0.8, water_tower));
    world.add(make_shared<sphere>(point3(7, 16, -7), 0.6, water_tower));
    
    // Decorative spheres (like sculptures or lights)
    auto decoration = make_shared<lambertian>(color(1.0, 0.8, 0.2));
    world.add(make_shared<sphere>(point3(0, 2, 0), 0.3, decoration));
    world.add(make_shared<sphere>(point3(-5, 1, 0), 0.2, decoration));
    world.add(make_shared<sphere>(point3(5, 1, 0), 0.2, decoration));

    camera cam;

    cam.aspect_ratio      = 16.0 / 9.0;
    cam.image_width       = 1200;
    cam.samples_per_pixel = 100;  // Good balance of quality and speed
    cam.max_depth         = 50;

    cam.vfov     = 45;  // Wider field of view to capture more of the city
    cam.lookfrom = point3(20, 20, 10);  // Elevated position to see the city
    cam.lookat   = point3(0, 8, 0);     // Look towards center of city
    cam.vup      = vec3(0,1,0);

    cam.defocus_angle = 0.2;  // Sharp focus for architectural details
    cam.focus_dist    = 25.0;




    // Start timing
    auto start_time = std::chrono::high_resolution_clock::now();


    //Main render call
    cam.render(world); 
    


    // Stop timing and calculate elapsed time
    auto end_time = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);
    std::cerr << "Rendering completed in " << duration.count() / 1000.0 << " seconds" << std::endl;
}
